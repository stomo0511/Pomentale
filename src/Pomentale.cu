#include "hip/hip_runtime.h"
/*
 ============================================================================
 Name        : Pomentale.cu
 Author      : stomo
 Version     :
 Copyright   : Your copyright notice
 Description : Compute sum of reciprocals using STL on CPU and Thrust on GPU
 ============================================================================
 */

#include <iostream>
#include <cstdlib>
#include <cmath>
#include <cassert>
#include <vector>
#include <algorithm>
#include <thrust/complex.h>

#define EPS 0.000001  // 停止判定
#define MAXIT 16      // 最大反復回数

int P;  // Pomentale法の次数

// ゼロ点
std::vector< thrust::complex<double> > Zrs
{
	thrust::complex<double> (  1.0,  1.0 ),  // z1
	thrust::complex<double> ( -1.0,  1.0 ),  // z2
	thrust::complex<double> (  0.0, -1.0 )   // z3
};

// ゼロ点の重複度
std::vector<double> Mul
{
	1.0,  // n1
	2.0,  // n2
	3.0   // n3
};

// Polynomial function value
template<typename T> thrust::complex<T> Pval( thrust::complex<T> z )
{
	thrust::complex<T> tmp;
	tmp = thrust::complex<T> (1.0,0.0);


	for (int i=0; i<Zrs.size(); i++)
	{
		tmp *= pow( z - Zrs[i], Mul[i] );
	}
	return tmp;
}

int main(int argc, char *argv[])
{
	if (argc<4)
	{
		std::cerr << "Usage: a.out [Order] [Real(z0)] [Imag(z0)]\n";
		exit(EXIT_FAILURE);
	}
	P = atoi(argv[1]); // Pomentale法の次数
	assert( (P==2) | (P==4) | (P==8) | (P==16) | (P==32) );

	double rez0 = atof(argv[2]);
	double imz0 = atof(argv[3]);

	thrust::complex<double> z0 = thrust::complex<double>( rez0, imz0 );
	thrust::complex<double> z = Pval(z0);

	std::cout << "z0 = (" << z0.real() << ", " << z0.imag() << ")\n";
	std::cout << "z  = (" << z.real() << ", " << z.imag() << ")\n";

	return 0;
}
